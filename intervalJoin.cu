#include "hip/hip_runtime.h"
// This program executes a typical Interval Join
#include <iostream>
#include <omp.h>
#include "intervalJoin.h"
using namespace std;

// This is the CPU version, please don't modify it
void intervalJoinCPU(int id)
{
    int i,j;
    #pragma omp parallel for    
    for(i=0;i<setB.length[id];i++){
        for(j=0;j<setA.length[id];j++){
            if((inStartA[j]<=inStartB[i] && inStartB[i]<=inEndA[j]) || (inStartA[j]<=inEndB[i] && inEndB[i]<=inEndA[j]) || (inStartA[j]<=inStartB[i] && inEndB[i]<=inEndB[j]) || (inStartA[j]>=inStartB[i] && inEndB[i]>=inEndA[j])){
                if(outCPU_Begin[i]==INT_MAX)
                    outCPU_Begin[i]=j;
                outCPU_End[i]=j;
            }
        }  
    }
    int total_intersects=0;
    for(i=0;i<setB.length[id];i++){
        if(outCPU_Begin[i]<INT_MAX && outCPU_End[i]>INT_MIN)
            total_intersects+=(outCPU_End[i]-outCPU_Begin[i]+1);
    }
	
	printf("Intersection count=%d\n",total_intersects);	
}

/***	Implement your CUDA Kernel here	***/
__global__
void intervalJoinGPU()
{
}
/***	Implement your CUDA Kernel here	***/

int main()
{
	int i;
	timespec time_begin, time_end;
	int intervalJoinCPUExecTime, intervalJoinGPUExecTime;
	int cpuTotalTime=0,gpuTotalTime=0; 
	FILE *fpA, *fpB;
	read_Meta();
	
	fpA = fopen ("data/dataA.csv","r");
	fpB = fopen ("data/dataB.csv","r");
	
	for(i=0;i<setA.count;i++){
		init_from_csv(fpA, fpB, i);
		
		clock_gettime(CLOCK_REALTIME, &time_begin);
		intervalJoinCPU(i);
		clock_gettime(CLOCK_REALTIME, &time_end);
		intervalJoinCPUExecTime = timespec_diff_us(time_begin, time_end);
		cout << "CPU time for executing a typical Interval Join = " <<  intervalJoinCPUExecTime / 1000 << "ms" << endl;
		cpuTotalTime+=intervalJoinCPUExecTime;
		
		clock_gettime(CLOCK_REALTIME, &time_begin);
		/***Do the required GPU Memory allocation here***/
		
		/***Do the required GPU Memory allocation here***/
		
		/***Configure the CUDA Kernel call here***/
		intervalJoinGPU<<<1,1>>>(); // Lunch the kernel
		
		hipDeviceSynchronize(); // Do synchronization before clock_gettime()
		
		/***Copy back the result from GPU Memory to CPU memory arrays outGPU_Begin and outGPU_End***/
		
		/***Copy back the result from GPU Memory to CPU memory arrays outGPU_Begin and outGPU_End***/
		
		clock_gettime(CLOCK_REALTIME, &time_end);
		intervalJoinGPUExecTime = timespec_diff_us(time_begin, time_end);
		cout << "GPU time for executing a typical Interval Join = " << intervalJoinGPUExecTime / 1000 << "ms" << endl;
		cpuTotalTime+=intervalJoinGPUExecTime;
		
		/*
		if(checker(setB.length[i])){
			cout << "Congratulations! You pass the check." << endl;
			cout << "Speedup: " << (float)intervalJoinCPUExecTime / intervalJoinGPUExecTime << endl;
		}
		else
			cout << "Sorry! Your result is wrong." << endl;
		*/
		ending();
		
	}
	
	fclose(fpA);
	fclose(fpB);

	return 0;
}
