#include "hip/hip_runtime.h"
// This program executes a typical Interval Join
#include <iostream>
#include "intervalJoin.h"
using namespace std;

// This is the CPU version, please don't modify it
void intervalJoinCPU()
{
	
}

/***	Implement your CUDA Kernel here	***/
__global__
void intervalJoinGPU()
{
}
/***	Implement your CUDA Kernel here	***/

int main()
{
	int intervalJoinCPUExecTime, intervalJoinGPUExecTime;
	read_Meta();
	init();	
	timespec time_begin, time_end;                                                 
	clock_gettime(CLOCK_REALTIME, &time_begin);
	intervalJoinCPU();
	clock_gettime(CLOCK_REALTIME, &time_end);
	intervalJoinCPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "CPU time for executing a typical Interval Join = " 
			 <<  intervalJoinCPUExecTime / 1000 << "ms" << endl;

	clock_gettime(CLOCK_REALTIME, &time_begin);
	/***	Lunch your CUDA Kernel here	***/

	intervalJoinGPU<<<1,1>>>(); // Lunch the kernel
	
	hipDeviceSynchronize(); // Do synchronization before clock_gettime()
	/***	Lunch your CUDA Kernel here	***/
	clock_gettime(CLOCK_REALTIME, &time_end);
	intervalJoinGPUExecTime = timespec_diff_us(time_begin, time_end);
	cout << "GPU time for executing a typical Interval Join = " 
			 << intervalJoinGPUExecTime / 1000 << "ms" << endl;

	if(checker()){
		cout << "Congratulations! You pass the check." << endl;
		cout << "Speedup: " << (float)intervalJoinCPUExecTime / intervalJoinGPUExecTime << endl;
	}
	else
		cout << "Sorry! Your result is wrong." << endl;

	ending();
	
	return 0;
}
